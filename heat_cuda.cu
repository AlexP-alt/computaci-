#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define BMP_HEADER_SIZE 54
#define ALPHA 0.01      //Thermal diffusivity
#define L 0.2           // Length (m) of the square domain
#define DX 0.02         // grid spacing in x-direction
#define DY 0.02         // grid spacing in y-direction
#define DT 0.0005       // Time step
#define T 1500          //Temperature on ºk of the heat source
//#define BLOCK_X 16
//#define BLOCK_Y 16

// Function to print the grid (optional, for debugging or visualization)
void print_grid(double *grid, int nx, int ny) {
    int i,j;
    for (i = 0; i < nx; i++) {
        for (j = 0; j < ny; j++) {
            printf("%.2f ", grid[i * ny + j]);
        }
        printf("\n");
    }
    printf("\n");
}
// Function to initialize the grid
void initialize_grid(double *grid, int nx, int ny,int temp_source) {
    int i,j;
    for(i = 0; i < nx; i++) {
        for (j = 0; j < ny; j++) {
           if(i==j) grid[i * ny + j] = 1500.0;
           else if(i== nx-1-j) grid[i * ny + j]=1500.0;
           else grid[i * ny + j]=0.0;
        }
    }
 
    }

__global__ void update_heat_kernel(double *grid,double *new_grid,int nx,int ny,double r){
    int i = blockIdx.y*blockDim.y+threadIdx.y;
    int j = blockIdx.x*blockDim.x+threadIdx.x;
    if(i>=nx || j>=ny) return;
    int idx=i*ny+j;
    if(i==0 || i==nx-1 || j==0 || j==ny-1){
        new_grid[idx]=0.0;
    }else{
        new_grid[idx]= grid[idx] +
                      r*(grid[(i + 1) * ny + j] + grid[(i - 1) * ny + j] - 2.0*grid[idx]) +
                      r*(grid[i * ny + (j + 1)] + grid[i * ny + (j - 1)] - 2.0*grid[idx]);
    }
}

void solve_heat_equation_cuda(double *h_grid,int steps,double r,int nx,int ny, int block_x, int block_y){
    size_t bytes = nx*ny*sizeof(double);
    double *d_grid,*d_new;
    hipMalloc(&d_grid,bytes);
    hipMalloc(&d_new,bytes);
    hipMemcpy(d_grid,h_grid,bytes,hipMemcpyHostToDevice);

    dim3 block(block_x,block_y);
    dim3 gridDim((ny+block.x-1)/block.x,(nx+block.y-1)/block.y);

    for(int step=0;step<steps;step++){
        update_heat_kernel<<<gridDim,block>>>(d_grid,d_new,nx,ny,r);
        hipDeviceSynchronize();
        double *temp=d_grid;
        d_grid=d_new;
        d_new=temp;
    }
    hipMemcpy(h_grid,d_grid,bytes,hipMemcpyDeviceToHost);
    hipFree(d_grid);
    hipFree(d_new);
}

// Function to write BMP file header
void write_bmp_header(FILE *file, int width, int height) {
    unsigned char header[BMP_HEADER_SIZE] = {0};

    int file_size = BMP_HEADER_SIZE + 3 * width * height;
    header[0] = 'B';
    header[1] = 'M';
    header[2] = file_size & 0xFF;
    header[3] = (file_size >> 8) & 0xFF;
    header[4] = (file_size >> 16) & 0xFF;
    header[5] = (file_size >> 24) & 0xFF;
    header[10] = BMP_HEADER_SIZE;

    header[14] = 40;  // Info header size
    header[18] = width & 0xFF;
    header[19] = (width >> 8) & 0xFF;
    header[20] = (width >> 16) & 0xFF;
    header[21] = (width >> 24) & 0xFF;
    header[22] = height & 0xFF;
    header[23] = (height >> 8) & 0xFF;
    header[24] = (height >> 16) & 0xFF;
    header[25] = (height >> 24) & 0xFF;
    header[26] = 1;   // Planes
    header[28] = 24;  // Bits per pixel

    fwrite(header, 1, BMP_HEADER_SIZE, file);
}

void get_color(double value, unsigned char *r, unsigned char *g, unsigned char *b) {
    
    if (value >= 500.0) {
        *r = 255; *g = 0; *b = 0; // Red
    } else if (value >= 100.0) {
        *r = 255; *g = 128; *b = 0; // Orange
    } else if (value >= 50.0) {
        *r = 171; *g = 71; *b = 188; // Lilac
    } else if(value>=25){
        *r = 255; *g = 255; *b = 0; // Yellow
    }else if (value >= 1) {
    *r = 0; *g = 0; *b = 255; // Blue
    }
    else if (value >= 0.1) {
        *r = 5; *g = 248; *b = 252; // Cyan
        }
    else{
    *r = 255; *g = 255; *b = 255; // white
    }
}
//Function to write the grid matrix into the file
void write_grid(FILE *file, double *grid,int nx,int ny)
{
    int i,j,padding;
    // Write pixel data to BMP file
    for (i = nx - 1; i >= 0; i--) { // BMP format stores pixels bottom-to-top
        for (j = 0; j < ny; j++) {
                unsigned char r, g, b;
                get_color(grid[i*ny+j], &r, &g, &b);
                fwrite(&b, 1, 1, file); // Write blue channel
                fwrite(&g, 1, 1, file); // Write green channel
                fwrite(&r, 1, 1, file); // Write red channel
            }
            // Row padding for 4-byte alignment (if necessary)
            for (padding = 0; padding < (4 - (nx * 3) % 4) % 4; padding++) {
                fputc(0, file);
            }
        }
}

// Main function
int main(int argc, char *argv[]) {
    clock_t time_begin, time_end;
    double r; // constant of the heat equation
    int nx,ny;  // Grid size in x-direction and y-direction
    int steps; // Number of time steps
    //double DT;
    if (argc!=5)
    {
        printf("Command line wrong\n");
        printf("Command line should be: heat_serial size steps name_output_file.bmp. block_size\n");
        printf("Try again!!!!\n");
        return 1;
    }
    int block_x, block_y;
    block_x=block_y=atoi(argv[4]);
    nx=ny=atoi(argv[1]);
    r= ALPHA * DT / (DX * DY);
    steps=atoi(argv[2]);
    time_begin=clock();
    // Allocate memory for the grid
    double *grid = (double *)calloc(nx * ny, sizeof(double));
    double *new_grid = (double *)calloc(nx * ny, sizeof(double));

    // Initialize the grid
    initialize_grid(grid, nx, ny, T);

    // Solve heat equation
    solve_heat_equation_cuda(grid,steps,r,nx, ny, block_x, block_y);
    // Write grid into a bmp file
    FILE *file = fopen(argv[3], "wb");
    if (!file) {
            printf("Error opening the output file.\n");
            return 1;
    }

    write_bmp_header(file, nx, ny);
    write_grid(file,grid,nx,ny);

    fclose(file);
    //Function to visualize the values of the temperature. Use only for debugging
    // print_grid(grid, nx, ny);
    // Free allocated memory
    free(grid);
    free(new_grid);
    time_end=clock();
    printf("The Execution Time=%fs with a matrix size of %dx%d and %d steps\n",(time_end-time_begin)/(double)CLOCKS_PER_SEC,nx,nx,steps);
    return 0;
}
